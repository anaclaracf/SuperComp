#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstring>
#include <cmath>
#include <string>
#include <tuple>
#include <fstream>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include<omp.h>

using namespace std;

vector<string> genSub(int size, string seq){
    vector<string> sub_seqs;
    for(int i=0; i< size; i++){
        string stg = "";
        for(int j=i; j < size; j++){
            stg += seq[j];
            sub_seqs.push_back(stg);
        }
    }

    return sub_seqs;
}


struct temp {
    thrust::device_ptr<char> sequence_A;
    char letter_B;
    thrust::device_ptr<int> line;

    temp(thrust::device_ptr<char> sequence_A_, char letter_B_, thrust::device_ptr<int> line_):
    sequence_A(sequence_A_),
    letter_B(letter_B_),
    line(line_){};

    __host__ __device__
    int operator()(const int& x){

        int diagonal = line[x-1] + (letter_B == sequence_A[x-1] ? 2 : -1);
        int deletion = line[x] - 1;

        int max = 0;

        if (deletion >= diagonal && deletion > max){
            return deletion;
        }
        else if (diagonal > max){
            return diagonal;
        }
        else{
            return max;
        }

    }

};

struct lateral {

    __host__ __device__
    int operator()(const int& i, const int& j){

        int max = 0;
        int actual = j;        
        int left = i-1;

        max = (left < actual ? actual : left);

        if(max < 0){
            max =  0;
        }

        return max;
    }

};


int main(){

    int n, m;
    string seq_A, seq_B;
    vector<string> sub_A, sub_B;
    thrust::device_vector<int> results[2], maximos;

    cin >> n >> m;
    cin >> seq_A >> seq_B;

    //  Gerar todas as subsequencias a´ e b´ não-nulas de a e b
    sub_A = genSub(n, seq_A);
    sub_B = genSub(m, seq_B);

    int size_A = sub_A.size();
    int size_B = sub_B.size();

    // criando contador inicial para percorrer a sequencia A
    thrust::counting_iterator<int> counter_initial(1);

    // utilizando openmp para paralelizar o processo
    #pragma omp for collapse(2)

    // percorre o tamanho das possíveis sequencias de A
    for (int i=0; i < size_A; i++) {
        // percorre o tamanho das possíveis sequencias de B
        for (int j=0; j < size_B; j++) {

            // transforma a sub sequencia em um device vector para utilização da GPU
            thrust::device_vector<char> subseq_A(sub_A[i].begin(), sub_A[i].end());
            thrust::device_vector<char> subseq_B(sub_B[j].begin(), sub_B[j].end());
                        
            int size_subB = int(subseq_B.size());
            int size_subA = int(subseq_A.size());

            thrust::counting_iterator<int> counter_final(size_subA+1);

            results[0].resize(size_subA+1);
            results[1].resize(size_subA+1);

            thrust::fill(results[0].begin(), results[0].end(),0);


            for(int e = 0; e < size_subB; e++){
                char letter_B = subseq_B[e];
                thrust::transform(counter_initial, counter_final, results[1].begin() + 1, temp(subseq_A.data(), letter_B, results[0].data()));
                thrust::inclusive_scan(results[1].begin()+1, results[1].end(), results[0].begin()+1, lateral());
            }

            maximos.push_back(results[0].back());

            // cout << results[0].back() << endl;
            
        }
        
    }

    int max_value = thrust::reduce(maximos.begin(), maximos.end(),0,thrust::maximum<int>());

    cout << max_value << endl;

    return 0;

}